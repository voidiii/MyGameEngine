#include "hip/hip_runtime.h"
#include "MGEpch.h"
#include "CUDAHead.cuh"

__global__ void ElasticCollisions_Verlet_GPU(const glm::vec2* a, const glm::vec2* b, glm::vec2* c, int N)
{
    int tid =  threadIdx.x;
    
    // Boundary check
    if (tid < N) {
        // Each thread adds a single element
        glm::vec2 hit_distance = a[tid] - b[tid];
    
        if (glm::length(hit_distance) < 0.0001f)
        {
            c[tid] = glm::vec2(0);
            return;
        };
    
        glm::vec2 hit_direction = glm::normalize(hit_distance);
    
        c[tid] = (1.0f - glm::length(hit_distance)) / 2.0f * hit_direction;
    }
}

void ElasticCollisions_Verlet_GPU_Jumper(const glm::vec2* a, const glm::vec2* b, glm::vec2* c, int N)
{
    ElasticCollisions_Verlet_GPU<< <1, 256>> >(a, b, c, N);
}