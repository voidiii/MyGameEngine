#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <chrono>

using std::cout;
using std::generate;
using std::vector;
const int SHMEM_SIZE = 1 << 10;
const int N = 1 << 10;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

/*
    int main()
    {
        const int arraySize = 5;
        const int a[arraySize] = { 1, 2, 3, 4, 5 };
        const int b[arraySize] = { 10, 20, 30, 40, 50 };
        int c[arraySize] = { 0 };

        // Add vectors in parallel.
        hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addWithCuda failed!");
            return 1;
        }

        printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
            c[0], c[1], c[2], c[3], c[4]);

        // hipDeviceReset must be called before exiting in order for profiling and
        // tracing tools such as Nsight and Visual Profiler to show complete traces.
        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return 1;
        }

        return 0;
    }
*/

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;
    int id = hipGetDevice(&id);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

__global__ void matrixMul(const int* a, const int* b, int* c, int N) {
    // Compute each thread's global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate over row, and down column
    c[row * N + col] = 0;
    for (int k = 0; k < N; k++) {
        // Accumulate results for a single element
        c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
}

__global__ void matrixMul_Shared(const int *a, const int *b, int *c) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Statically allocated shared memory
  __shared__ int s_a[SHMEM_SIZE];
  __shared__ int s_b[SHMEM_SIZE];

  // Accumulate in temporary variable
  int tmp = 0;

  // Sweep tile across matrix
  for (int i = 0; i < N; i += blockDim.x) {
    // Load in elements for this tile
    s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * N + i + threadIdx.x];
    s_b[threadIdx.y * blockDim.x + threadIdx.x] =
        b[i * N + threadIdx.y * N + col];

    // Wait for both tiles to be loaded in before doing computation
    __syncthreads();

    // Do matrix multiplication on the small matrix
    for (int j = 0; j < blockDim.x; j++) {
      tmp +=
          s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
    }

    // Wait for all threads to finish using current tiles before loading in new
    // ones
    __syncthreads();
  }

  // Write back results
  c[row * N + col] = tmp;
}

// Check result on the CPU
void verify_result(vector<int>& a, vector<int>& b, vector<int>& c, int N) {
    // For every row...
    for (int i = 0; i < N; i++) {
        // For every column...
        for (int j = 0; j < N; j++) {
            // For every element in the row-column pair
            int tmp = 0;
            for (int k = 0; k < N; k++) {
                // Accumulate the partial results
                tmp += a[i * N + k] * b[k * N + j];
            }

            // Check against the CPU result
            assert(tmp == c[i * N + j]);
        }
    }
}

int main() {
    // Matrix size of 1024 x 1024;
    int N = 1 << 10;

    // Size (in bytes) of matrix
    size_t bytes = N * N * sizeof(int);

    // Host vectors
    vector<int> h_a(N * N);
    vector<int> h_b(N * N);
    vector<int> h_c(N * N);

    // Initialize matrices
    generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
    generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

    // Allocate device memory
    int* d_a, * d_b, * d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Copy data to the device
    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    // Threads per CTA dimension
    int THREADS = 32;

    // Blocks per grid dimension (assumes THREADS divides N evenly)
    int BLOCKS = N / THREADS;

    // Use dim3 structs for block  and grid dimensions
    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    auto start = std::chrono::high_resolution_clock::now();

    // Launch kernel
    matrixMul << <blocks, threads >> > (d_a, d_b, d_c, N);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    cout << "GPU took:" << duration.count() << "\n";
    // Copy back to the host
    hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    start = std::chrono::high_resolution_clock::now();

    // Check result
    verify_result(h_a, h_b, h_c, N);

    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    cout << "CPU took:" << duration.count() << "\n";

    cout << "COMPLETED SUCCESSFULLY\n";
    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
